#include <iostream>
#include <hip/hip_runtime.h>

// https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__TYPES.html
#define CHECK(val) check_cuda( (val), #val, __FILE__, __LINE__ )
void check_cuda(hipError_t res, const char *func, const char *file, const int line) {
	if (!res)
		return ;
	std::cerr << "CUDA error = " << static_cast<unsigned int>(res);
	std::cerr << " at " << file << ":" << line << " '" << func << "' \n";
	hipDeviceReset();
	exit(1);
}

__global__ void matrix_mul_kernel(float *M, float *N, float *P, int side) {
	int i = threadIdx.x;
	int j = threadIdx.y;
	float sum = 0;
	for (int k = 0; k < side; ++k) {
		float m = M[i * side + k];
		float n = N[k * side + j];
		sum += m * n;
	}
	P[i * side + j] = sum;
}

void matrix_mul(float *M, float *N, float *P, int side) {
	unsigned long long size = side * side * sizeof(float);
	float *d_M, *d_N, *d_P;
	CHECK(hipMalloc(&d_M, size));
	CHECK(hipMalloc(&d_N, size));
	CHECK(hipMalloc(&d_P, size));
	CHECK(hipMemcpy(d_M, M, size, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_N, N, size, hipMemcpyHostToDevice));

	dim3 blocks(1, 1, 1);
	dim3 threads(side, side, 1);
	matrix_mul_kernel<<<blocks, threads>>>(d_M, d_N, d_P, side);
	CHECK(hipGetLastError());
	CHECK(hipDeviceSynchronize());
	
	CHECK(hipMemcpy(P, d_P, size, hipMemcpyDeviceToHost));
	
	CHECK(hipFree(d_M));
	CHECK(hipFree(d_N));
	CHECK(hipFree(d_P));
}
