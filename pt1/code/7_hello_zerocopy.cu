#include <iostream>
#include <hip/hip_runtime.h>

// https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__TYPES.html
#define CHECK(val) check_cuda( (val), #val, __FILE__, __LINE__ )
void check_cuda(hipError_t res, const char *func, const char *file, const int line)
{
	if (!res)
		return ;
	std::cerr << "CUDA error = " << static_cast<unsigned int>(res);
	std::cerr << " at " << file << ":" << line << " '" << func << "' \n";
	hipDeviceReset();
	exit(1);
}

__global__ void	hello(const unsigned char *mem)
{
	unsigned int idx = threadIdx.z * blockDim.y * blockDim.x
					 + threadIdx.y * blockDim.x
					 + threadIdx.x;

	printf("I'm %u and got %c\n", idx, mem[idx]);
}

int main(int argc, char** argv)
{

	dim3	blocks;
	dim3	threads(2, 2, 2);

	unsigned char *mem;
	CHECK(hipHostMalloc((void **)&mem, 8 * sizeof(unsigned char), hipHostMallocDefault));
	CHECK(hipMemcpy(mem, "abcdefgh", 8 * sizeof(unsigned char), hipMemcpyHostToDevice));

	hello<<<blocks, threads>>>(mem);
	CHECK(hipGetLastError());
	CHECK(hipDeviceSynchronize());

	CHECK(hipHostFree(mem));
}
