#include <iostream>
#include <hip/hip_runtime.h>

// https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__TYPES.html
#define CHECK(val) check_cuda( (val), #val, __FILE__, __LINE__ )
void check_cuda(hipError_t res, const char *func, const char *file, const int line)
{
	if (!res)
		return ;
	std::cerr << "CUDA error = " << static_cast<unsigned int>(res);
	std::cerr << " at " << file << ":" << line << " '" << func << "' \n";
	hipDeviceReset();
	exit(1);
}

__global__ void	hello()
{
	unsigned int idx = threadIdx.z * blockDim.y * blockDim.x
					 + threadIdx.y * blockDim.x
					 + threadIdx.x;

	__shared__ unsigned int mem[8];
	mem[idx] = idx;
	__syncthreads();

	if (idx == 0) {
		printf("I'm first %u\n", idx);
	} else {
		printf("I'm %u and got %d\n", idx, mem[idx-1]);
	}
}

int main(int argc, char** argv)
{
	dim3	blocks;
	dim3	threads(2, 2, 2);

	hello<<<blocks, threads>>>();
	CHECK(hipGetLastError());
	CHECK(hipDeviceSynchronize());
}
