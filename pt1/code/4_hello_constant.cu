#include <iostream>
#include <hip/hip_runtime.h>

// https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__TYPES.html
#define CHECK(val) check_cuda( (val), #val, __FILE__, __LINE__ )
void check_cuda(hipError_t res, const char *func, const char *file, const int line)
{
	if (!res)
		return ;
	std::cerr << "CUDA error = " << static_cast<unsigned int>(res);
	std::cerr << " at " << file << ":" << line << " '" << func << "' \n";
	hipDeviceReset();
	exit(1);
}

__constant__ unsigned char mem[8];

__global__ void	hello()
{
	unsigned int idx = threadIdx.z * blockDim.y * blockDim.x
					 + threadIdx.y * blockDim.x
					 + threadIdx.x;

	printf("I'm %u and got %c\n", idx, mem[idx]);
}

int main(int argc, char** argv)
{
	dim3	blocks;
	dim3	threads(2, 2, 2);

	int offset = 0;
	CHECK(hipMemcpyToSymbol(HIP_SYMBOL(mem), "abcdefgh", 8 * sizeof(unsigned char), offset, hipMemcpyHostToDevice));

	hello<<<blocks, threads>>>();
	CHECK(hipGetLastError());
	CHECK(hipDeviceSynchronize());
}
