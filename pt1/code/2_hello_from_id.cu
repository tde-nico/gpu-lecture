#include <iostream>
#include <hip/hip_runtime.h>

// https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__TYPES.html
#define CHECK(val) check_cuda( (val), #val, __FILE__, __LINE__ )
void check_cuda(hipError_t res, const char *func, const char *file, const int line)
{
	if (!res)
		return ;
	std::cerr << "CUDA error = " << static_cast<unsigned int>(res);
	std::cerr << " at " << file << ":" << line << " '" << func << "' \n";
	hipDeviceReset();
	exit(1);
}

__global__ void	hello()
{
	unsigned int idx = threadIdx.z * blockDim.y * blockDim.x
					 + threadIdx.y * blockDim.x
					 + threadIdx.x;
	printf("Hello World from %u\n", idx);
}

int main(int argc, char** argv)
{
	dim3	blocks;
	dim3	threads(2, 2, 2);

	hello<<<blocks, threads>>>();
	CHECK(hipGetLastError());
	CHECK(hipDeviceSynchronize());
}
